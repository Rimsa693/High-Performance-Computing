#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o passwordcrack_cuda passwordcrack_cuda.cu
    ./passwordcrack_cuda
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char passr1[] = "RA1234";
	char passr2[] = "AN3029";
	char passr3[] = "GO1998";
	char passr4[] = "SA0023";


	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;
	char *d1 = passr1;
	char *d2 = passr2;
	char *d3 = passr3;
	char *d4 = passr4;

	while(*a == *d1) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",passr1);
			break;
		}

		a++;
		d1++;
	}
	
	while(*b == *d2) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",passr2);
			break;
		}

		b++;
		d2++;
	}

	while(*c == *d3) { 
		if(*c == '\0') 
		{
			printf("Password: %s\n",passr3);
			break;
		}

		c++;
		d3++;
	}

	while(*d == *d4) { 
		if(*d == '\0') 
		{
			printf("Password: %s\n",passr4);
			return 1;
		}

		d++;
		d4++;
	}
	return 0;

}

__global__ void  kernel() {
	char a1,a2,a3,a4;

	char passr[7];
	passr[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	passr[0] = firstMatch;
	passr[1] = secondMatch;
	for(a1='0'; a1<='9'; a1++){
		for(a2='0'; a2<='9'; a2++){
			for(a3='0'; a3<='9'; a3++){
				for(a4='0'; a4<='9'; a4++){
					passr[2] = a1;
					passr[3] = a2;
					passr[4] = a3;
					passr[5] = a4;
					 
					if(is_a_match(passr)) {
					} 
					else {
	     			//printf("tried: %s\n", passr);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int dsecond =  finish->tv_sec - start->tv_sec; 
	long long int dnsecond =  finish->tv_nsec - start->tv_nsec; 
	if(dnsecond < 0 ) {
		dsecond--;
		dnsecond += 1000000000; 
	} 
	*difference = dsecond * 1000000000 + dnsecond;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldnsecond or %0.9lfsecond\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


